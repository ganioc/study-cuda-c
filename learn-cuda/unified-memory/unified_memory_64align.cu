
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

#define STRIDE_64K  65536


__global__ void init(int n, float *x, float *y){
    int lane_id = threadIdx.x &31;
    size_t warp_id = (threadIdx.x + blockDim.x * blockIdx.x)>> 5;
    size_t warps_per_grid = (blockDim.x* gridDim.x) >>5;
    size_t warp_total = ((sizeof(float) *n) + STRIDE_64K-1) / STRIDE_64K;


//    int index = blockIdx.x * blockDim.x + threadIdx.x;
//    int stride = blockDim.x * gridDim.x;
//    for(int i=index; i< n; i+= stride){
//        x[i] = 1.0f;
//        y[i] = 2.0f;
//    }
    for(; warp_id < warp_total; warp_id += warps_per_grid){
        #pragma unroll
        for(int rep = 0; rep < STRIDE_64K/sizeof(float)/32; rep++){
            size_t ind = warp_id * STRIDE_64K/sizeof(float) + rep*32 + lane_id;
            if( ind < n){
                x[ind] = 1.0f;
                y[ind] = 2.0f;
            }
        }
    }
}

// CUDA kernel to add elements of two arrays
__global__ void add(int n, float *x, float *y){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i=index; i< n; i+= stride){
        y[i] = x[i] + y[i];
    }
}
int main(void){
    int N = 1<<20;
    float *x, *y;

    // Allocate Unified Memoy, from 
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
//    for(int i = 0; i< N; i++){
  //      x[i] = 1.0f;
  //      y[i] = 2.0f;
  //  }
    
    // Launch kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (N + blockSize -1)/ blockSize;
    init<<<numBlocks, blockSize>>>(N, x, y);
    add<<<numBlocks, blockSize>>>(N, x, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for errors (all values shouold be 3.0f)
    float maxError = 0.0f;
    for(int i = 0; i< N; i++){
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout <<"Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    return 0;

}


