#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define N 512

void host_add(int *a, int*b, int *c){
    for(int idx=0; idx< N; idx++){
        c[idx]=a[idx]+b[idx];
    }
}

void fill_array(int *data){
    for(int idx=0;idx<N; idx++){
        data[idx] = idx;
    }
}
void print_output(int *a, int *b, int*c){
    for(int idx=0; idx<N;idx++){
        printf("\n %d + %d = %d", a[idx], b[idx],c[idx]);
    }
}

__global__ void device_add(int* a, int* b, int* c){
    // c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
    // c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    c[index] = a[index] + b[index];
}
int main(void){
    int *a,*b,*c;
    int *d_a, *d_b, *d_c;
    int size = N * sizeof(int);
    int threads_per_block;
    int no_of_blocks ;


    a = (int *) malloc(size);
    fill_array(a);
    b = (int*)malloc(size);
    fill_array(b);
    c = (int*)malloc(size);

    hipMalloc((void**)&d_a , N*sizeof(int) );
    hipMalloc((void**)&d_b, N*sizeof(int));
    hipMalloc((void**)&d_c, N*sizeof(int));

    hipMemcpy(d_a,a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b,b, N*sizeof(int), hipMemcpyHostToDevice);

    threads_per_block = 8;
    no_of_blocks = N/threads_per_block;

    device_add<<<no_of_blocks, threads_per_block>>>(d_a, d_b, d_c);

    hipMemcpy( c, d_c , N*sizeof(int) ,  hipMemcpyDeviceToHost);

    // host_add(a,b,c);
    print_output(a,b, c);
    free(a);
    free(b);
    free(c);
    hipFree( d_a);
    hipFree( d_b);
    hipFree( d_c);

    return 0;
}